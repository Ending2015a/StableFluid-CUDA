#include "hip/hip_runtime.h"

#include "pcg_solver.hpp"

#include "../common/error_helper.hpp"


PCGsolver::PCGsolver(int max_iter, double tol)  : max_iter(max_iter), tolerance(tol)
{
    // initialize cuBLAS & cuSPARSE
    error_check(hipblasCreate(&cubHandle));
    error_check(hipsparseCreate(&cusHandle));

    // create descriptor of matrix A
    error_check(hipsparseCreateMatDescr(&descr_A));

    // initialize properties of matrix A
    error_check(hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
    error_check(hipsparseSetMatFillMode(descr_A, HIPSPARSE_FILL_MODE_LOWER));
    error_check(hipsparseSetMatDiagType(descr_A, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    error_check(hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO));

    // create descriptor of matrix L
    error_check(hipsparseCreateMatDescr(&descr_L));

    // initialize properties of matrix L
    error_check(hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
    error_check(hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO));
    error_check(hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER));
    error_check(hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT));
}

PCGsolver::~PCGsolver()
{
    // free data
    free_memory();

    // cusparse
    hipsparseDestroyMatDescr(descr_A);
    hipsparseDestroyMatDescr(descr_L);
    hipsparseDestroy(cusHandle);
    hipblasDestroy(cubHandle);
}

void PCGsolver::solve_gpumem(int N, int nz,
                        double *d_A, int *d_rowIdx, int *d_colIdx,
                        double *d_b, double *d_guess)
{
    // check size
    this->N = N;
    this->nonzero = nz;
    free_memory();
    check_and_resize();

    // analyze matrix A (This will be used in incomplete-cholesky factorization)
    cusparseSolveAnalysisInfo_t info_A;
    error_check(cusparseCreateSolveAnalysisInfo(&info_A));
    error_check(cusparseDcsrsv_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, nonzero, descr_A, d_A, d_rowIdx, d_colIdx, info_A));

    // copy matrix A
    error_check(hipMemcpy(d_ic, d_A, nonzero * sizeof(double), hipMemcpyDeviceToDevice));

    // compute IC factorization
    error_check(cusparseDcsric0(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, descr_A, d_ic, d_rowIdx, d_colIdx, info_A));

    // analyze matrix L & U
    cusparseSolveAnalysisInfo_t info_L;
    error_check(cusparseCreateSolveAnalysisInfo(&info_L));
    error_check(cusparseDcsrsv_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, nonzero, descr_L, d_ic, d_rowIdx, d_colIdx, info_L));

    cusparseSolveAnalysisInfo_t info_U;
    error_check(cusparseCreateSolveAnalysisInfo(&info_U));
    error_check(cusparseDcsrsv_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                            N, nonzero, descr_L, d_ic, d_rowIdx, d_colIdx, info_U));

    // set initial guess
    if(d_guess == NULL)
    {
        error_check(hipMemset(d_x, 0, N * sizeof(double)));
    }
    else
    {
        error_check(hipMemcpy(d_x, d_guess, N * sizeof(double), hipMemcpyDeviceToDevice));
    }
    error_check(hipMemcpy(d_r, d_b, N * sizeof(double), hipMemcpyDeviceToDevice));
    
    // solve
    int k;
    for(k=0;k<max_iter;++k)
    {
        error_check(hipblasDnrm2(cubHandle, N, d_r, 1, &rTr));
        if(rTr < tolerance)
           break;
        
        error_check(cusparseDcsrsv_solve(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            N, &alpha1, descr_L, d_ic, d_rowIdx, d_colIdx, info_L, d_r, d_y));

        error_check(cusparseDcsrsv_solve(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                            N, &alpha1, descr_L, d_ic, d_rowIdx, d_colIdx, info_U, d_y, d_z));

        rhot = rho;  //store last rho
        error_check(hipblasDdot(cubHandle, N, d_r, 1, d_z, 1, &rho));  //compute new rho
        
        if(k == 0)
        {
            error_check(hipblasDcopy(cubHandle, N, d_z, 1, d_p, 1));
        }
        else
        {
            beta = rho/rhot;
            error_check(hipblasDscal(cubHandle, N, &beta, d_p, 1));
            error_check(hipblasDaxpy(cubHandle, N, &alpha1, d_z, 1, d_p, 1));
        }
        // Compute q <- Ap
        error_check(hipsparseDcsrmv(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        N, N, nonzero, &alpha1, descr_A, d_A, d_rowIdx, d_colIdx, d_p, &beta0, d_q));

        error_check(hipblasDdot(cubHandle, N, d_p, 1, d_q, 1, &pTq));
        alpha = rho/pTq;
        error_check(hipblasDaxpy(cubHandle, N, &alpha, d_p, 1, d_x, 1));
        double nalpha = -alpha;
        error_check(hipblasDaxpy(cubHandle, N, &nalpha, d_q, 1, d_r, 1));
    }

    std::cout << "[PCGsolver] solved in " << k << " iterations, final norm(r) = " 
              << std::scientific << rTr << std::endl;

    error_check(cusparseDestroySolveAnalysisInfo(info_A));
    error_check(cusparseDestroySolveAnalysisInfo(info_L));
    error_check(cusparseDestroySolveAnalysisInfo(info_U));

}

void PCGsolver::convert_coo2csr(const int N, const int nonzero, const int* cooRowIdx, int *csrRowIdx)
{
    error_check(hipsparseXcoo2csr(cusHandle, cooRowIdx, nonzero, N, csrRowIdx, HIPSPARSE_INDEX_BASE_ZERO));
}

double *PCGsolver::get_device_x()
{
    return d_x;
}

void PCGsolver::free_memory()
{
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_r);
    hipFree(d_rt);
    hipFree(d_xt);
    hipFree(d_q);
    hipFree(d_p);
    hipFree(d_ic);
}

void PCGsolver::check_and_resize()
{
    error_check(hipMalloc(&d_x, N * sizeof(double)));
    error_check(hipMalloc(&d_y, N * sizeof(double)));
    error_check(hipMalloc(&d_z, N * sizeof(double)));
    error_check(hipMalloc(&d_r, N * sizeof(double)));
    error_check(hipMalloc(&d_rt, N * sizeof(double)));
    error_check(hipMalloc(&d_xt, N * sizeof(double)));
    error_check(hipMalloc(&d_q, N * sizeof(double)));
    error_check(hipMalloc(&d_p, N * sizeof(double)));
    d_N = N;
    error_check(hipMalloc(&d_ic, nonzero * sizeof(double)));
    d_nz = nonzero;
}
